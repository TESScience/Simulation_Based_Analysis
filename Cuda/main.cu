#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "random.h"
#include "cuda_utils.h"
 
int main( int argc, char* argv[] )
{
    // Size of vectors
    const int n = 100000;
    const size_t bytes = n*sizeof(double);
    // Number of threads in each thread block
    const int blockSize = 1024;
    // Number of thread blocks in grid
    const int gridSize = (int)ceil((float)n/blockSize);

    // Get the device count
    printf("Device Count: %i\n", cuda_device_count() );

    int i;
    for(i = 0; i < cuda_device_count(); i++ )
        printf("Warp Size for CUDA device %i: %i\n", i, warp_size(i));
 
    // Host input vectors
    double *h_a = (double*) alloca(bytes);
    double *h_b = (double*) alloca(bytes);
    //Host output vector
    double *h_c =  (double*) alloca(bytes);
 
    // Device input vectors
    double *d_a, *d_b;
    //Device output vector
    double *d_c;
 
 
    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
 
    // Initialize vectors on host
    for(i = 0; i < n; i++ ) {
        h_a[i] = sin(i)*sin(i);
        h_b[i] = cos(i)*cos(i);
    }
 
    // Copy host vectors to device
    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);
 
 
    // Execute the kernel
    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
 
    // Copy array back to host
    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );
 
    // Sum up vector c and print result divided by n, this should equal 1 within error
    double sum = 0;
    for(i=0; i<n; i++)
        sum += h_c[i];
    printf("final result: %f\n", sum/n);
 
    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}
