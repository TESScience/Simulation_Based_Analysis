#include "hip/hip_runtime.h"
#include "random.h"
#include "warpspeed.h"

__global__ void vectorAdd(const float *A, const float *B, float *C)
{
    const int id = threadIdx.x + 
                   threadIdx.y * blockDim.x + 
                   threadIdx.z * blockDim.x * blockDim.y;

    C[id] = A[id] + B[id];
}
 
__global__ void warpspeed_rand(uint32_t * random_data)
{
    const int id = threadIdx.x + 
                   threadIdx.y * blockDim.x + 
                   threadIdx.z * blockDim.x * blockDim.y;
    //const int warpid = id / warpSize;
    uint32_t temp;

    temp = random_data[id];
    temp += random_data[id + TAP1 - RANDOM_FRAME_SIZE];
    temp += random_data[id + TAP2 - RANDOM_FRAME_SIZE];
    temp += random_data[id + TAP3 - RANDOM_FRAME_SIZE];
    temp += random_data[id + LENGTH - RANDOM_FRAME_SIZE];
    random_data[id] = temp;
}
