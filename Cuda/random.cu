#include "hip/hip_runtime.h"
#include "random.h"
#include "warpspeed.h"
 
__global__ void warpspeed_rand(uint32_t * random_data)
{
    // Get our global thread ID
    int id = threadIdx.x + 
              threadIdx.y * blockDim.x + 
              threadIdx.z * blockDim.x * blockDim.y;
    uint32_t temp;

    temp = random_data[id];
    temp += random_data[id + TAP1 - RANDOM_FRAME_SIZE];
    temp += random_data[id + TAP2 - RANDOM_FRAME_SIZE];
    temp += random_data[id + TAP3 - RANDOM_FRAME_SIZE];
    temp += random_data[id + LENGTH - RANDOM_FRAME_SIZE];
    random_data[id] = temp;
}

// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(double *a, double *b, double *c, int n)
{
    // int id = blockIdx.x*blockDim.x + threadIdx.x;

    // Get our global thread ID
    int id = threadIdx.x + 
             threadIdx.y * blockDim.x + 
             threadIdx.z * blockDim.x * blockDim.y;

    //int warpid = id / 32;
 
    // Make sure we do not go out of bounds
    if (id < n)
        c[id] = a[id] + b[id];
}
