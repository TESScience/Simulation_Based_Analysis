#include "cuda_utils.h"
#include <assert.h>
#include <hip/hip_runtime.h>

/**
 * Get the device count for CUDA boards.
 */
int cuda_device_count() {
   int nDevices;
   hipGetDeviceCount(&nDevices);
   return nDevices;
}

/** 
 * Get the maximum number of threads per block
 */
int max_threads_per_block(const int device_idx) {
   assert(device_idx < cuda_device_count());
   hipDeviceProp_t prop;
   hipGetDeviceProperties(&prop, device_idx);
   return prop.maxThreadsPerBlock;
}
