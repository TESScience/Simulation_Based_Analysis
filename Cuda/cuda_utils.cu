#include "cuda_utils.h"
#include <assert.h>

/**
 * Get the device count for CUDA boards.
 */
int cuda_device_count() {
   int nDevices;
   hipGetDeviceCount(&nDevices);
   return nDevices;
}

int warp_size(const int i) {
   assert(i < cuda_device_count());
   hipDeviceProp_t prop;
   hipGetDeviceProperties(&prop, i);
   return prop.warpSize;
}
