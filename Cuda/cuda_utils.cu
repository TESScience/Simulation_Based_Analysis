
#include <hip/hip_runtime.h>
/**
 * Get the device count for CUDA boards.
 */
int cuda_device_count() {
   int nDevices;
   hipGetDeviceCount(&nDevices);
   return nDevices;
}
